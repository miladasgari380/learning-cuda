
#include <hip/hip_runtime.h>
#include <stdio.h>

void initWith(float num, float* a, int N){

	for(int i=0; i<N; ++i)
		a[i] = num;

}

__global__
void addVectorsInto(float* results, float* a, float* b,int N){

	int withinGrid = threadIdx.x + blockIdx.x*blockDim.x;
	int gridStrid = gridDim.x * blockDim.x;

	for(int i=withinGrid; i<N; i+=gridStrid)
		results[i] = a[i] + b[i];
}

void checkElementsAre(float target, float *array, int N){
  
	for(int i = 0; i < N; i++){
		if(array[i] != target){
			printf("FAIL: array[%d] - %0.0f does not equal %0.0f\n", i, array[i], target);
			exit(1);
		}
	}
	printf("SUCCESS! All values added correctly.\n");
}

int main(){

	const int N = 2 << 20;
	size_t size = N * sizeof(float);
	float* a, *b, *c;

	hipMallocManaged(&a, size);
	hipMallocManaged(&b, size);
	hipMallocManaged(&c, size);

	initWith(3, a, N);
	initWith(4, b, N);
	initWith(0, c, N);

	addVectorsInto<<<32, 1024>>>(c, a, b, N);
	hipDeviceSynchronize();

	checkElementsAre(7, c, N);

	hipFree(a);
	hipFree(b);
	hipFree(c);

	return 0;
}
